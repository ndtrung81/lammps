#include "hip/hip_runtime.h"
// **************************************************************************
//                                   edpd.cu
//                             -------------------
//                           Trung Dac Nguyen (U Chicago)
//
//  Device code for acceleration of the edpd pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                : September 2023
//    email                : ndactrung@gmail.com
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)
#include "lal_aux_fun1.h"
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( vel_tex,float4);
#else
_texture_2d( pos_tex,int4);
_texture_2d( vel_tex,int4);
#endif
#else
#define pos_tex x_
#define vel_tex v_
#endif

#define EPSILON (numtyp)1.0e-10

//#define _USE_UNIFORM_SARU_LCG
//#define _USE_UNIFORM_SARU_TEA8
//#define _USE_GAUSSIAN_SARU_LCG

#if !defined(_USE_UNIFORM_SARU_LCG) && !defined(_USE_UNIFORM_SARU_TEA8) && !defined(_USE_GAUSSIAN_SARU_LCG)
#define _USE_UNIFORM_SARU_LCG
#endif

// References:
// 1. Y. Afshar, F. Schmid, A. Pishevar, S. Worley, Comput. Phys. Comm. 184 (2013), 1119–1128.
// 2. C. L. Phillips, J. A. Anderson, S. C. Glotzer, Comput. Phys. Comm. 230 (2011), 7191-7201.
// PRNG period = 3666320093*2^32 ~ 2^64 ~ 10^19

#define LCGA 0x4beb5d59 /* Full period 32 bit LCG */
#define LCGC 0x2600e1f7
#define oWeylPeriod 0xda879add /* Prime period 3666320093 */
#define oWeylOffset 0x8009d14b
#define TWO_N32 0.232830643653869628906250e-9f /* 2^-32 */

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0]
// using the inherent LCG, then multiply u with sqrt(3) to "match"
// with a normal random distribution.
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
// Curly brackets to make variables local to the scope.
#ifdef _USE_UNIFORM_SARU_LCG
#define SQRT3 (numtyp)1.7320508075688772935274463
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  state = LCGA*state + LCGC;                                                  \
  wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod);   \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns uniformly distributed random numbers u in [-1.0;1.0] using TEA8
// then multiply u with sqrt(3) to "match" with a normal random distribution
// Afshar et al. mutlplies u in [-0.5;0.5] with sqrt(12)
#ifdef _USE_UNIFORM_SARU_TEA8
#define SQRT3 (numtyp)1.7320508075688772935274463
#define k0 0xA341316C
#define k1 0xC8013EA4
#define k2 0xAD90777D
#define k3 0x7E95761E
#define delta 0x9e3779b9
#define rounds 8
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));         \
  unsigned int wstate = (state + seed2) ^ (((signed int)state)>>8);           \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int sum = 0;                                                       \
  for (int i=0; i < rounds; i++) {                                            \
    sum += delta;                                                             \
    state += ((wstate<<4) + k0)^(wstate + sum)^((wstate>>5) + k1);            \
    wstate += ((state<<4) + k2)^(state + sum)^((state>>5) + k3);              \
  }                                                                           \
  unsigned int v = (state ^ (state>>26)) + wstate;                            \
  unsigned int s = (signed int)((v^(v>>20))*0x6957f5a7);                      \
  randnum = SQRT3*(s*TWO_N32*(numtyp)2.0-(numtyp)1.0);                        \
}
#endif

// specifically implemented for steps = 1; high = 1.0; low = -1.0
// returns two uniformly distributed random numbers r1 and r2 in [-1.0;1.0],
// and uses the polar method (Marsaglia's) to transform to a normal random value
// This is used to compared with CPU DPD using RandMars::gaussian()
#ifdef _USE_GAUSSIAN_SARU_LCG
#define saru(seed1, seed2, seed, timestep, randnum) {                         \
  unsigned int seed3 = seed + timestep;                                       \
  seed3^=(seed1<<7)^(seed2>>6);                                               \
  seed2+=(seed1>>4)^(seed3>>15);                                              \
  seed1^=(seed2<<9)+(seed3<<8);                                               \
  seed3^=0xA5366B4D*((seed2>>11) ^ (seed1<<1));                               \
  seed2+=0x72BE1579*((seed1<<4)  ^ (seed3>>16));                              \
  seed1^=0x3F38A6ED*((seed3>>5)  ^ (((signed int)seed2)>>22));                \
  seed2+=seed1*seed3;                                                         \
  seed1+=seed3 ^ (seed2>>2);                                                  \
  seed2^=((signed int)seed2)>>17;                                             \
  unsigned int state=0x12345678;                                              \
  unsigned int wstate=12345678;                                               \
  state  = 0x79dedea3*(seed1^(((signed int)seed1)>>14));                      \
  wstate = (state + seed2) ^ (((signed int)state)>>8);                        \
  state  = state + (wstate*(wstate^0xdddf97f5));                              \
  wstate = 0xABCB96F7 + (wstate>>1);                                          \
  unsigned int v, s;                                                          \
  numtyp r1, r2, rsq;                                                         \
  while (1) {                                                                 \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r1 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    state = LCGA*state + LCGC;                                                \
    wstate = wstate + oWeylOffset+((((signed int)wstate)>>31) & oWeylPeriod); \
    v = (state ^ (state>>26)) + wstate;                                       \
    s = (signed int)((v^(v>>20))*0x6957f5a7);                                 \
    r2 = s*TWO_N32*(numtyp)2.0-(numtyp)1.0;                                   \
    rsq = r1 * r1 + r2 * r2;                                                  \
    if (rsq < (numtyp)1.0) break;                                             \
  }                                                                           \
  numtyp fac = ucl_sqrt((numtyp)-2.0*log(rsq)/rsq);                           \
  randnum = r2*fac;                                                           \
}
#endif

// note the change in coeff: coeff.x = a0, coeff.y = gamma, coeff.z = cut (no sigma)

__kernel void k_edpd(const __global numtyp4 *restrict x_,
                     const __global numtyp4 *restrict extra,
                     const __global numtyp4 *restrict coeff,
                     const __global numtyp4 *restrict coeff2,
                     const __global numtyp4 *restrict sc,
                     const __global numtyp4 *restrict kc,
                     const int lj_types,
                     const __global numtyp *restrict sp_lj,
                     const __global numtyp *restrict sp_sqrt,
                     const __global int * dev_nbor,
                     const __global int * dev_packed,
                     __global acctyp3 *restrict ans,
                     __global acctyp *restrict engv,
                     __global acctyp *restrict Q,
                     const int eflag, const int vflag, const int inum,
                     const int nbor_pitch,
                     const __global numtyp4 *restrict v_,
                     const __global numtyp *restrict cutsq,
                     const numtyp dtinvsqrt, const int seed,
                     const int timestep, const int tstat_only,
                     const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    int itype=ix.w;
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp4 Tcvi = extra[i];
    numtyp Ti = Tcvi.x;
    numtyp cvi = Tcvi.y;

    numtyp factor_dpd, factor_sqrt;
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      factor_dpd = sp_lj[sbmask(j)];
      factor_sqrt = sp_sqrt[sbmask(j)];
      j &= NEIGHMASK;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      int jtype=jx.w;
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      int mtype=itype*lj_types+jtype;
      if (rsq<cutsq[mtype]) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;
        
        const numtyp coeffx=coeff[mtype].x;
        const numtyp coeffy=coeff[mtype].y;
        const numtyp coeffz=coeff[mtype].z;
        const numtyp coeffw=coeff[mtype].w; // cut[itype][jtype]

        const numtyp4 Tcvj = extra[j];
        numtyp Tj = Tcvj.x;
        numtyp cvj = Tcvj.y;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }

        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        numtyp T_ij=(numtyp)0.5*(Ti+Tj);
        numtyp4 T_pow;
        T_pow.x = T_ij - (numtyp)1.0;
        T_pow.y = T_pow.x*T_pow.y;
        T_pow.z = T_pow.x*T_pow.z;
        T_pow.w = T_pow.x*T_pow.w;

        numtyp power_d = coeff2[mtype].x; // power[itype][jtype]
        if (1) { // power_flag
          numtyp factor = (numtyp)1.0;
          factor += sc[mtype].x*T_pow.x + 
                    sc[mtype].y*T_pow.y +
                    sc[mtype].z*T_pow.z +
                    sc[mtype].w*T_pow.w;
          power_d *= factor;
        }

        power_d = MAX((numtyp)0.01,power_d);
        numtyp wc = (numtyp)1.0 - r/coeffw; // cut[itype][jtype]
        wc = MAX((numtyp)0.0,MIN((numtyp)1.0,wc));
        numtyp wr = ucl_pow(wc, (numtyp)0.5*power_d);

        numtyp kboltz = (numtyp)1.0;
        numtyp GammaIJ = coeff[mtype].y; // gamma[itype][jtype]
        numtyp SigmaIJ = (numtyp)4.0*GammaIJ*kboltz*Ti*Tj/(Ti+Tj);
        SigmaIJ = ucl_sqrt(SigmaIJ);

        numtyp force =  coeff[mtype].x*T_ij*wc; // a0[itype][jtype]
        force += SigmaIJ * wr *randnum * dtinvsqrt;
        force *= factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // heat transfer
        numtyp dQc,dQd,dQr;
        numtyp coeff2x = coeff2[mtype].x; //power[itype][jtype]
        numtyp coeff2y = coeff2[mtype].y; //kappa[itype][jtype]
        numtyp coeff2z = coeff2[mtype].z; //powerT[itype][jtype]
        numtyp coeff2w = coeff2[mtype].w; //cutT[itype][jtype]
        if (r < coeff2w) {  
          numtyp wrT = 1.0 - r/coeff2w;
          wrT = MAX(0.0,MIN(1.0,wrT));
          wrT = pow(wrT, 0.5*coeff2z); // powerT[itype][jtype]
          numtyp randnumT = saru(tag1, tag2, seed, timestep, randnum); // randomT->gaussian();
          randnumT = MAX(-5.0,MIN(randnum,5.0));

          numtyp kappaT = coeff2y; // kappa[itype][jtype]
          if (1) { // kappa_flag
            numtyp factor = (numtyp)1.0;
            factor += kc[mtype].x*T_pow.x +
                      kc[mtype].y*T_pow.y +
                      kc[mtype].z*T_pow.z +
                      kc[mtype].w*T_pow.w;
            kappaT *= factor;
          }

          numtyp kij = cvi*cvj*kappaT * T_ij*T_ij;
          numtyp alphaij = ucl_sqrt((numtyp)2.0*kboltz*kij);

          dQc  = kij * wrT*wrT * (Tj - Ti )/(Ti*Tj);
          dQd  = wr*wr*( GammaIJ * vijeij*vijeij - SigmaIJ*SigmaIJ/mass[itype] ) - SigmaIJ * wr *vijeij *randnum;
          dQd /= (cvi+cvj);
          dQr  = alphaij * wrT * dtinvsqrt * randnumT;
          Q[i] += (dQc + dQd + dQr );
        }

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0.5*T_ij*coeffx*coeffw * wc*wc;
          energy+=factor_dpd*e;
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

__kernel void k_edpd_fast(const __global numtyp4 *restrict x_,
                          const __global numtyp4 *restrict extra,
                          const __global numtyp4 *restrict coeff_in,
                          const __global numtyp4 *restrict coeff2_in,
                          const __global numtyp4 *restrict sc_in,
                          const __global numtyp4 *restrict kc_in,
                          const __global numtyp *restrict sp_lj_in,
                          const __global numtyp *restrict sp_sqrt_in,
                          const __global int * dev_nbor,
                          const __global int * dev_packed,
                          __global acctyp3 *restrict ans,
                          __global acctyp *restrict engv,
                          __global acctyp *restrict Q,
                          const int eflag, const int vflag, const int inum,
                          const int nbor_pitch,
                          const __global numtyp4 *restrict v_,
                          const __global numtyp *restrict cutsq,
                          const numtyp dtinvsqrt, const int seed,
                          const int timestep, const int tstat_only,
                          const int t_per_atom) {
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  #ifndef ONETYPE
  __local numtyp4 coeff[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp4 coeff2[MAX_SHARED_TYPES*MAX_SHARED_TYPES];
  __local numtyp sp_lj[4];
  __local numtyp sp_sqrt[4];
  if (tid<4) {
    sp_lj[tid]=sp_lj_in[tid];
    sp_sqrt[tid]=sp_sqrt_in[tid];
  }
  if (tid<MAX_SHARED_TYPES*MAX_SHARED_TYPES) {
    coeff[tid]=coeff_in[tid];
    coeff2[tid]=coeff2_in[tid];
    sc[tid]=sc_in[tid];
    kc[tid]=kc_in[tid];
  }
  __syncthreads();
  #else
  const numtyp coeffx=coeff_in[ONETYPE].x;
  const numtyp coeffy=coeff_in[ONETYPE].y;
  const numtyp coeffz=coeff_in[ONETYPE].z;
  const numtyp coeffw=coeff_in[ONETYPE].w;
  const numtyp cutsq_p=cutsq[ONETYPE];
  #endif

  int n_stride;
  local_allocate_store_pair();

  acctyp3 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    for (int i=0; i<6; i++) virial[i]=(acctyp)0;
  }

  if (ii<inum) {
    int i, numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];
    #ifndef ONETYPE
    int iw=ix.w;
    int itype=fast_mul((int)MAX_SHARED_TYPES,iw);
    #endif
    numtyp4 iv; fetch4(iv,i,vel_tex); //v_[i];
    int itag=iv.w;

    const numtyp4 Tcvi = extra[i];
    numtyp Ti = Tcvi.x;
    numtyp cvi = Tcvi.y;

    #ifndef ONETYPE
    numtyp factor_dpd, factor_sqrt;
    #endif
    for ( ; nbor<nbor_end; nbor+=n_stride) {
      ucl_prefetch(dev_packed+nbor+n_stride);

      int j=dev_packed[nbor];
      #ifndef ONETYPE
      factor_dpd = sp_lj[sbmask(j)];
      factor_sqrt = sp_sqrt[sbmask(j)];
      j &= NEIGHMASK;
      #endif

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];
      #ifndef ONETYPE
      int mtype=itype+jx.w;
      const numtyp cutsq_p=cutsq[mtype];
      #endif
      numtyp4 jv; fetch4(jv,j,vel_tex); //v_[j];
      int jtag=jv.w;

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<cutsq_p) {
        numtyp r=ucl_sqrt(rsq);
        if (r < EPSILON) continue;

        numtyp rinv=ucl_recip(r);
        numtyp delvx = iv.x - jv.x;
        numtyp delvy = iv.y - jv.y;
        numtyp delvz = iv.z - jv.z;
        numtyp dot = delx*delvx + dely*delvy + delz*delvz;

        #ifndef ONETYPE
        const numtyp coeffx=coeff[mtype].x;
        const numtyp coeffy=coeff[mtype].y;
        const numtyp coeffz=coeff[mtype].z;
        #endif
        const numtyp coeffw=coeff[mtype].w; // cut[itype][jtype]

        const numtyp4 Tcvj = extra[j];
        numtyp Tj = Tcvj.x;
        numtyp cvj = Tcvj.y;

        unsigned int tag1=itag, tag2=jtag;
        if (tag1 > tag2) {
          tag1 = jtag; tag2 = itag;
        }
        numtyp randnum = (numtyp)0.0;
        saru(tag1, tag2, seed, timestep, randnum);

        numtyp T_ij=(numtyp)0.5*(Ti+Tj);
        numtyp4 T_pow;
        T_pow.x = T_ij - (numtyp)1.0;
        T_pow.y = T_pow.x*T_pow.y;
        T_pow.z = T_pow.x*T_pow.z;
        T_pow.w = T_pow.x*T_pow.w;

        numtyp power_d = coeff2[mtype].x; // power[itype][jtype]
        if (1) { // power_flag
          numtyp factor = (numtyp)1.0;
          factor += sc[mtype].x*T_pow.x + 
                    sc[mtype].y*T_pow.y +
                    sc[mtype].z*T_pow.z +
                    sc[mtype].w*T_pow.w;
          power_d *= factor;
        }

        power_d = MAX((numtyp)0.01,power_d);
        numtyp wc = (numtyp)1.0 - r/coeffw; // cut[itype][jtype]
        wc = MAX((numtyp)0.0,MIN((numtyp)1.0,wc));
        numtyp wr = ucl_pow(wc, (numtyp)0.5*power_d);

        numtyp kboltz = (numtyp)1.0;
        numtyp GammaIJ = coeff[mtype].y; // gamma[itype][jtype]
        numtyp SigmaIJ = (numtyp)4.0*GammaIJ*kboltz*Ti*Tj/(Ti+Tj);
        SigmaIJ = ucl_sqrt(SigmaIJ);

        numtyp force =  coeff[mtype].x*T_ij*wc; // a0[itype][jtype]
        force += SigmaIJ * wr *randnum * dtinvsqrt;
        force *= factor_dpd*rinv;

        f.x+=delx*force;
        f.y+=dely*force;
        f.z+=delz*force;

        // heat transfer
        numtyp dQc,dQd,dQr;
        numtyp coeff2x = coeff2[mtype].x; //power[itype][jtype]
        numtyp coeff2y = coeff2[mtype].y; //kappa[itype][jtype]
        numtyp coeff2z = coeff2[mtype].z; //powerT[itype][jtype]
        numtyp coeff2w = coeff2[mtype].w; //cutT[itype][jtype]
        if (r < coeff2w) {  
          numtyp wrT = 1.0 - r/coeff2w;
          wrT = MAX(0.0,MIN(1.0,wrT));
          wrT = pow(wrT, 0.5*coeff2z); // powerT[itype][jtype]
          numtyp randnumT = saru(tag1, tag2, seed, timestep, randnum); // randomT->gaussian();
          randnumT = MAX(-5.0,MIN(randnum,5.0));

          numtyp kappaT = coeff2y; // kappa[itype][jtype]
          if (1) { // kappa_flag
            numtyp factor = (numtyp)1.0;
            factor += kc[mtype].x*T_pow.x +
                      kc[mtype].y*T_pow.y +
                      kc[mtype].z*T_pow.z +
                      kc[mtype].w*T_pow.w;
            kappaT *= factor;
          }

          numtyp kij = cvi*cvj*kappaT * T_ij*T_ij;
          numtyp alphaij = ucl_sqrt((numtyp)2.0*kboltz*kij);

          dQc  = kij * wrT*wrT * (Tj - Ti )/(Ti*Tj);
          dQd  = wr*wr*( GammaIJ * vijeij*vijeij - SigmaIJ*SigmaIJ/mass[itype] ) - SigmaIJ * wr *vijeij *randnum;
          dQd /= (cvi+cvj);
          dQr  = alphaij * wrT * dtinvsqrt * randnumT;
          Q[i] += (dQc + dQd + dQr );
        }

        if (EVFLAG && eflag) {
          numtyp e = (numtyp)0.5*T_ij*coeffx*coeffw * wc*wc;
          #ifndef ONETYPE
          energy+=factor_dpd*e;
          #else
          energy+=e;
          #endif
        }
        if (EVFLAG && vflag) {
          virial[0] += delx*delx*force;
          virial[1] += dely*dely*force;
          virial[2] += delz*delz*force;
          virial[3] += delx*dely*force;
          virial[4] += delx*delz*force;
          virial[5] += dely*delz*force;
        }
      }

    } // for nbor
  } // if ii
  store_answers(f,energy,virial,ii,inum,tid,t_per_atom,offset,eflag,vflag,
                ans,engv);
}

